#include "Util.h"

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
  CHECK(hipGetDeviceProperties(&devProv, 0));
  printf("**********GPU info**********\n");
  printf("Name: %s\n", devProv.name);
  printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
  printf("Num SMs: %d\n", devProv.multiProcessorCount);
  printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
  printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
  printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
  printf("CMEM: %lu bytes\n", devProv.totalConstMem);
  printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
  printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);
  printf("****************************\n");
}